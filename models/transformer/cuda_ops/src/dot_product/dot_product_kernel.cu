#include "hip/hip_runtime.h"
#include "../cuda_utils.h"
#include "dot_product_kernel.h"


__global__ void dot_product_forward_kernel(
    int m, int h, int kkk, int c, const float* query, const float* pos, float* out_F, const int* kq_map
)
{
    // m: # of total mappings
    // h: # of attention heads
    // kkk: # of keys (kernel volume)
    // c: # of attention channels
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= m * h) return;

    int map_idx = index / h;
    int head_idx = index % h;

    int query_idx_ = kq_map[m + map_idx]; // kq_map[1][map_idx]
    int kernel_idx = kq_map[map_idx] % kkk;

    for(int i = 0; i < c; i++){

        int query_idx = query_idx_ * h * c + head_idx * c + i;
        int pos_idx = kernel_idx * h * c + head_idx * c + i;

        out_F[index] += query[query_idx] * pos[pos_idx];
    }
}

__global__ void dot_product_backward_kernel(
    int m, int h, int kkk, int c, const float* query, const float* pos, const int* kq_map,
    float* grad_query, float* grad_pos, const float* grad_out_F
)
{
    // m: # of total mappings
    // h: # of attention heads
    // kkk: # of keys (kernel volume)
    // c: # of attention channels
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= m * c) return;

    int map_idx = index / c;
    int i = index % c;

    int query_idx_ = kq_map[m + map_idx]; // kq_map[1][map_idx]
    int kernel_idx = kq_map[map_idx] % kkk;

    for(int head_idx = 0; head_idx < h; head_idx++){

        int out_F_idx = map_idx * h + head_idx;
        int query_idx = query_idx_ * h * c + head_idx * c + i;
        int pos_idx = kernel_idx * h * c + head_idx * c + i;

        atomicAdd(grad_query + query_idx, grad_out_F[out_F_idx] * pos[pos_idx]);
        atomicAdd(grad_pos + pos_idx, grad_out_F[out_F_idx] * query[query_idx]);
    }
}

void dot_product_forward_launcher(
    int m, int h, int kkk, int c, const float* query, const float* pos, float* out_F, const int* kq_map
) {
    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    dim3 blocks(DIVUP(m * h, THREADS_PER_BLOCK));
    dim3 threads(THREADS_PER_BLOCK);
    dot_product_forward_kernel<<<blocks, threads, 0, stream>>>(
        m, h, kkk, c, query, pos, out_F, kq_map
    );
}

void dot_product_backward_launcher(
    int m, int h, int kkk, int c, const float* query, const float* pos, const int* kq_map,
    float* grad_query, float* grad_pos, const float* grad_out_F
) {
    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    dim3 blocks(DIVUP(m * c, THREADS_PER_BLOCK));
    dim3 threads(THREADS_PER_BLOCK);
    dot_product_backward_kernel<<<blocks, threads, 0, stream>>>(
        m, h, kkk, c, query, pos, kq_map,
        grad_query, grad_pos, grad_out_F
    );
}
